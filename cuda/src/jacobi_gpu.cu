#include "hip/hip_runtime.h"
#include "jacobi_gpu.h"
#include "timestamp.h"
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define THREADS_PER_BLOCK 256 // Also check with other values, e.g. 512.

// TODO: For greatly improving the per-thread error sum-reduction process inside each block, we can
//       follow this optimization guide (1st link is slides, 2nd link is video explanation):
//       - https://developer.download.nvidia.com/assets/cuda/files/reduction.pdf
//       - https://www.youtube.com/watch?v=bpbit8SPMxU

////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void jacobi_iteration_gpu(
    const double *d_src, double *d_dst,
    int maxXCount, int maxYCount,
    double xStart, double yStart,
    double deltaX, double deltaY,
    double alpha, double omega,
    double *d_errors)
{
    int by = blockIdx.y;
    int bx = blockIdx.x;
    int bi = by * gridDim.x + bx;

    int ty = threadIdx.y;
    int tx = threadIdx.x;
    int ti = ty * blockDim.x + tx;

    int y = blockIdx.y * blockDim.y + threadIdx.y + 1;
    int x = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int i = y * maxXCount + x;
    bool isValidIndex = y < maxYCount-1 && x < maxXCount-1;

    double thisCell = d_src[i];

    __shared__ double cache[THREADS_PER_BLOCK]; // L1 cache.

    // Each thread will do one global/L2 memory access (except threads with out-of-bounds indices).
    if (isValidIndex)
        cache[ti] = thisCell;

    __syncthreads();

    double error = 0.0;

    if (isValidIndex) {
        // Corner cells will do two global/L2 reads and two L1 reads.
        // The rest border cells will do one global/L2 reads and three L1 reads.
        // All non-border cells will do four L1 reads.
        double leftCell  = tx == 0            ? d_src[i-1]         : cache[ti-1];
        double rightCell = tx == blockDim.x-1 ? d_src[i+1]         : cache[ti+1];
        double upperCell = ty == 0            ? d_src[i-maxXCount] : cache[ti-blockDim.x];
        double belowCell = ty == blockDim.y-1 ? d_src[i+maxXCount] : cache[ti+blockDim.x];

        // Coefficients
        // TODO: Is it better to always recalculate them OR precalculate them in global memory and
        //       then always accessing global memory or L2 cache?
        double cx = 1.0/(deltaX*deltaX);
        double cy = 1.0/(deltaY*deltaY);
        double cc = -2.0*cx-2.0*cy-alpha;

        // TODO: If we precalculate any of these values, then each thread will require at least one
        //       global/L2 read. Therefore, it's probably better to simply always recalculate them (?).
        double fY = yStart + y*deltaY;
        double fX = xStart + x*deltaX;
        double f = -alpha*(1.0-fX*fX)*(1.0-fY*fY) - 2.0*(1.0-fX*fX) - 2.0*(1.0-fY*fY);

        double updateVal = (
            (leftCell + rightCell) * cx +
                (upperCell + belowCell) * cy +
                thisCell * cc -f
        ) / cc;

        d_dst[i] = thisCell - omega*updateVal;
        error = updateVal*updateVal;
    }

    // Finally, do an iteration of sum-reduction since we already have the per-thread errors easily
    // accessible. The rest of the reduction will be done by repeated executions of the
    // "sum_reduction" kernel.

    // Cache is no longer needed for storing matrix values. So we can use it for the sum reduction.
    cache[ti] = error;

    __syncthreads();

    for (unsigned int s = 1; s < THREADS_PER_BLOCK; s <<= 1) {
        if (ti % (s << 1) == 0)
            cache[ti] += cache[ti + s];
        __syncthreads();
    }

    if (ti == 0)
        d_errors[bi] = cache[0];
}

__global__ void sum_reduction(double *d_errors, int stride, int maxThreadIdx)
{
    int bx = blockIdx.x;
    int tx = threadIdx.x;
    int ti = threadIdx.y * blockDim.x + tx;

    __shared__ double cache[THREADS_PER_BLOCK];

    cache[tx] = d_errors[ti];

    __syncthreads();

    for (unsigned int s = 1; s < THREADS_PER_BLOCK; s <<= 1) {
        if (tx % (s << 1) == 0)
            cache[tx] += cache[tx + s];
        __syncthreads();
    }

    if (tx == 0)
        d_errors[bx] = cache[0];
}

////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////////////////

struct pair_t {
    int first;
    int second;
};

extern "C" pair_t findMostCloseToSqrtDivisor(int number)
{
    pair_t divisors = {1, number};

    for(int i = ceil(sqrt(number)); i > 0; i--)
        if(number % i == 0) {
            divisors.first = i;
            divisors.second = number/i;
            break;
        }

    return divisors;
}

////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////////////////

extern "C" float jacobi_gpu(
    double *src, double *dst,
    int maxXCount, int maxYCount,
    double xStart, double yStart,
    double deltaX, double deltaY,
    double alpha, double omega,
    int max_iteration_count, double max_acceptable_error,
    int *out_iteration_count, double *out_error, float *out_elapsedTime)
{
    double *d_src, *d_dst;
    double *d_tb_errors; // thread-block errors.

    ////////////////////////////////////////////////////////////////////////////////////////////////
    ///

    pair_t blockSideSizes = findMostCloseToSqrtDivisor(THREADS_PER_BLOCK);
    dim3 blockSize(blockSideSizes.first, blockSideSizes.second);

    int blocksCount = ceil(((maxXCount-2)*(maxYCount-2)) / THREADS_PER_BLOCK);
    pair_t gridSideSizes = findMostCloseToSqrtDivisor(blocksCount);
    dim3 gridSize(gridSideSizes.first, blockSideSizes.second);

    ////////////////////////////////////////////////////////////////////////////////////////////////
    /// Allocate device memory & fill it with host memory data.

    hipError_t err;
    size_t bytesCnt = maxXCount * maxYCount * sizeof(double);

    // Allocate device memory.
    err = hipMalloc((void **) &d_src, bytesCnt);
    if (err != hipSuccess) {
        fprintf(stderr, "GPUassert: %s\n", err);
        return err;
    }
    err = hipMalloc((void **) &d_dst, bytesCnt);
    if (err != hipSuccess) {
        fprintf(stderr, "GPUassert: %s\n", err);
        return err;
    }
    err = hipMalloc((void **) &d_tb_errors, blocksCount * sizeof(double));
    if (err != hipSuccess) {
        fprintf(stderr, "GPUassert: %s\n", err);
        return err;
    }

    // Copy data to device memory.
    err = hipMemcpy(d_src, src, bytesCnt, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "GPUassert: %s\n", err);
        return err;
    }
    err = hipMemcpy(d_dst, dst, bytesCnt, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "GPUassert: %s\n", err);
        return err;
    }

    ////////////////////////////////////////////////////////////////////////////////////////////////
    ///

    timestamp t_start;
    t_start = getTimestamp();

    double error = HUGE_VAL;
    int iteration_count = -1;

    while (++iteration_count < max_iteration_count && err > max_acceptable_error)
    {
        jacobi_iteration_gpu<<<gridSize, blockSize>>>(
            d_src, d_dst,
            maxXCount, maxYCount,
            xStart, yStart,
            deltaX, deltaY,
            alpha, omega,
            d_tb_errors);

        // Do a sum-reduce across all per-blocks errors.
        // TODO: For sure there is some error in the logic of either the following loop, or the
        //       "sum_reduction" function or both. But without a running environment to test, it is
        //       not so easy.
        int errBlocksCount = blocksCount;
        for (int err_stride = 1; ; err_stride *= THREADS_PER_BLOCK) {
            int maxThreadIdx = errBlocksCount;
            errBlocksCount = ceil(errBlocksCount / THREADS_PER_BLOCK);
            sum_reduction<<<errBlocksCount, THREADS_PER_BLOCK>>>(d_tb_errors, err_stride, maxThreadIdx);
            if (errBlocksCount == 1)
                break;
        }

        err = sqrt(d_tb_errors[0]) / ((maxXCount-2)*(maxYCount-2));

        // Swap buffers.
        double *tmp = d_src;
        d_src = d_dst;
        d_dst = tmp;
    }

    float elapsedTime = getElapsedtime(t_start);

    ////////////////////////////////////////////////////////////////////////////////////////////////
    /// Copy result back to host memory & deallocate device memory.

    // Copy results back to host memory
    err = hipMemcpy(dst, d_dst, bytesCnt, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "GPUassert: %s\n", err);
        return err;
    }

    err = hipFree(d_dst);
    if (err != hipSuccess) {
        fprintf(stderr, "GPUassert: %s\n", err);
        return err;
    }
    err = hipFree(d_src);
    if (err != hipSuccess) {
        fprintf(stderr, "GPUassert: %s\n", err);
        return err;
    }
    err = hipFree(d_tb_errors);
    if (err != hipSuccess) {
        fprintf(stderr, "GPUassert: %s\n", err);
        return err;
    }

    ////////////////////////////////////////////////////////////////////////////////////////////////
    /// Set function output values and return ok.

    *out_elapsedTime = elapsedTime;
    *out_iteration_count = iteration_count;
    *out_error = error;

    return 0.f;
}
