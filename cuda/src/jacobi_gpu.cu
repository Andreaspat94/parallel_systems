#include "hip/hip_runtime.h"
#include "jacobi_gpu.h"
#include "timestamp.h"
#include <math.h>
#include <hip/hip_runtime.h>

extern "C" double jacobi_iteration_gpu(
    const double *src, double *dst,
    int maxXCount, int maxYCount,
    double xStart, double yStart,
    double deltaX, double deltaY,
    double alpha, double omega)
{
#define SRC(XX,YY) src[(YY)*maxXCount+(XX)]
#define DST(XX,YY) dst[(YY)*maxXCount+(XX)]

    // Coefficients
    double cx = 1.0/(deltaX*deltaX);
    double cy = 1.0/(deltaY*deltaY);
    double cc = -2.0*cx-2.0*cy-alpha;

    double error = 0.0;
    double updateVal;

    for (int y = 1; y < (maxYCount-1); y++)
    {
        double fY = yStart + (y-1)*deltaY;
        for (int x = 1; x < (maxXCount-1); x++)
        {
            double fX = xStart + (x-1)*deltaX;
            double f = -alpha*(1.0-fX*fX)*(1.0-fY*fY) - 2.0*(1.0-fX*fX) - 2.0*(1.0-fY*fY);
            updateVal = (
                (SRC(x-1,y) + SRC(x+1,y))*cx +
                (SRC(x,y-1) + SRC(x,y+1))*cy +
                SRC(x,y)*cc - f
            ) / cc;
            DST(x,y) = SRC(x,y) - omega*updateVal;
            error += updateVal*updateVal;
        }
    }

    return sqrt(error)/((maxXCount-2)*(maxYCount-2));
}

extern "C" float jacobi_gpu(
    double *src, double *dst,
    int maxXCount, int maxYCount,
    double xStart, double yStart,
    double deltaX, double deltaY,
    double alpha, double omega,
    int max_iteration_count, double max_acceptable_error,
    int *out_iteration_count, double *out_error, float *out_elapsedTime)
{
    float *d_src, *d_dst;
    hipError_t err;

    ////////////////////////////////////////////////////////////////////////////////////////////////
    /// Allocate device memory & fill it with host memory data.

    size_t bytesCnt = maxXCount * maxYCount * sizeof(float);

    // Allocate device memory.
    err = hipMalloc((void **) &d_src, bytesCnt);
    if (err != hipSuccess) {
        fprintf(stderr, "GPUassert: %s\n", err);
        return err;
    }
    err = hipMalloc((void **) &d_dst, bytesCnt);
    if (err != hipSuccess) {
        fprintf(stderr, "GPUassert: %s\n", err);
        return err;
    }

    // Copy data to device memory.
    err = hipMemcpy(d_src, src, bytesCnt, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "GPUassert: %s\n", err);
        return err;
    }
    err = hipMemcpy(d_dst, dst, bytesCnt, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "GPUassert: %s\n", err);
        return err;
    }

    ////////////////////////////////////////////////////////////////////////////////////////////////
    /// Run the jacobi iterations.

    double error = HUGE_VAL;
    int iteration_count = -1;

    timestamp t_start;
    t_start = getTimestamp();

    while (++iteration_count < max_iteration_count && err > max_acceptable_error)
    {
        error = jacobi_iteration_gpu(
            src, dst, maxXCount, maxYCount, xStart, yStart, deltaX, deltaY, alpha, omega);

        // Swap buffers.
        double *temp = src; src = dst; dst = temp;
    }

    float elapsedTime = getElapsedtime(t_start);

    ////////////////////////////////////////////////////////////////////////////////////////////////
    /// Copy result back to host memory & deallocate device memory.

    // Copy results back to host memory
    err = hipMemcpy(dst, d_dst, bytesCnt, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "GPUassert: %s\n", err);
        return err;
    }

    err = hipFree(d_dst);
    if (err != hipSuccess) {
        fprintf(stderr, "GPUassert: %s\n", err);
        return err;
    }
    err = hipFree(d_src);
    if (err != hipSuccess) {
        fprintf(stderr, "GPUassert: %s\n", err);
        return err;
    }

    ////////////////////////////////////////////////////////////////////////////////////////////////
    /// Set function output values and return ok.

    *out_elapsedTime = elapsedTime;
    *out_iteration_count = iteration_count;
    *out_error = error;

    return 0.f;
}
